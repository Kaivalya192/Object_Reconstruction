#include "hip/hip_runtime.h"

// #include "stdafx.h"
#include "common.h"
#include "CUDASolverBundling.h"
#include "cudaUtil.h"
#include "../CUDACache.h"
#define THREADS_PER_BLOCK 512  // keep consistent with the GPU

extern "C" void evalMaxResidual(SolverInput& input, SolverState& state, SolverStateAnalysis& analysis, SolverParameters& parameters, CUDATimer* timer);
// extern "C" void buildVariablesToCorrespondencesTableCUDA(EntryJ* d_correspondences, unsigned int numberOfCorrespondences, unsigned int maxNumCorrespondencesPerImage, int* d_variablesToCorrespondences, int* d_numEntriesPerRow, CUDATimer* timer);
extern "C" void buildVariablesToCorrespondencesTableCUDA(EntryJ* d_correspondences, unsigned int numberOfCorrespondences, unsigned int maxNumCorrespondencesPerImage, int* d_variablesToCorrespondences, int* d_numEntriesPerRow, CUDATimer* timer, int *d_n_match_per_pair, const int n_images);
extern "C" void solveBundlingStub(SolverInput& input, SolverState& state, SolverParameters& parameters, SolverStateAnalysis& analysis, float* convergenceAnalysis, CUDATimer* timer);

extern "C" int countHighResiduals(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer* timer);

extern "C" void convertLiePosesToMatricesCU(const float3* d_rot, const float3* d_trans, unsigned int numTransforms, float4x4* d_transforms, float4x4* d_transformInvs);
extern "C" void collectHighResiduals(SolverInput& input, SolverState& state, SolverStateAnalysis& analysis, SolverParameters& parameters, CUDATimer* timer);
extern "C" void VisualizeCorrespondences(const uint2& imageIndices, const SolverInput& input, SolverState& state, SolverParameters& parameters, float3* d_corrImage);

#ifdef DEBUG_PRINT_SPARSE_RESIDUALS
extern "C" float EvalResidual(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer* timer);
#endif

CUDASolverBundling::CUDASolverBundling(unsigned int maxNumberOfImages, unsigned int maxNumResiduals, const int max_corr_per_image, const std::vector<int> &update_pose_flags, std::shared_ptr<YAML::Node> yml1)
	: m_maxNumberOfImages(maxNumberOfImages)
{
	yml = yml1;
	m_timer = NULL;
	//m_timer = new CUDATimer();
	//if (GlobalBundlingState::get().s_enableDetailedTimings) m_timer = new CUDATimer();
	m_bRecordConvergence = false;

	//TODO PARAMS
	// const unsigned int submapSize = GlobalBundlingState::get().s_submapSize;
	const unsigned int submapSize = 20;

	m_verifyOptDistThresh = 0.02f;//GlobalAppState::get().s_verifyOptDistThresh;
	m_verifyOptPercentThresh = 0.05f;//GlobalAppState::get().s_verifyOptPercentThresh;

	const unsigned int numberOfVariables = maxNumberOfImages;
	m_maxCorrPerImage = max_corr_per_image;

	printf("maxNumResiduals / maxNumberOfImages = %d / %d = %d\n", maxNumResiduals, maxNumberOfImages, int(maxNumResiduals/maxNumberOfImages));
	printf("m_maxNumberOfImages*m_maxCorrPerImage = %d x %d = %d\n", m_maxNumberOfImages, m_maxCorrPerImage, m_maxNumberOfImages*m_maxCorrPerImage);

#ifdef DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

	// State
	hipMalloc(&m_solverState.d_deltaRot, sizeof(float3)*numberOfVariables);
	hipMalloc(&m_solverState.d_deltaTrans, sizeof(float3)*numberOfVariables);
	hipMalloc(&m_solverState.d_rRot, sizeof(float3)*numberOfVariables);
	hipMalloc(&m_solverState.d_rTrans, sizeof(float3)*numberOfVariables);
	hipMalloc(&m_solverState.d_zRot, sizeof(float3)*numberOfVariables);
	hipMalloc(&m_solverState.d_zTrans, sizeof(float3)*numberOfVariables);
	hipMalloc(&m_solverState.d_pRot, sizeof(float3)*numberOfVariables);
	hipMalloc(&m_solverState.d_pTrans, sizeof(float3)*numberOfVariables);
	hipMalloc(&m_solverState.d_Jp, sizeof(float3)*maxNumResiduals);
	hipMalloc(&m_solverState.d_Ap_XRot, sizeof(float3)*numberOfVariables);
	hipMalloc(&m_solverState.d_Ap_XTrans, sizeof(float3)*numberOfVariables);
	hipMalloc(&m_solverState.d_scanAlpha, sizeof(float) * 2);
	hipMalloc(&m_solverState.d_rDotzOld, sizeof(float) *numberOfVariables);
	hipMalloc(&m_solverState.d_precondionerRot, sizeof(float3)*numberOfVariables);
	hipMalloc(&m_solverState.d_precondionerTrans, sizeof(float3)*numberOfVariables);
	hipMalloc(&m_solverState.d_sumResidual, sizeof(float));
	unsigned int n = (maxNumResiduals + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
	hipMalloc(&m_solverExtra.d_maxResidual, sizeof(float) * n);
	hipMalloc(&m_solverExtra.d_maxResidualIndex, sizeof(int) * n);
	m_solverExtra.h_maxResidual = new float[n];
	m_solverExtra.h_maxResidualIndex = new int[n];

	hipMalloc(&d_variablesToCorrespondences, sizeof(int)*m_maxNumberOfImages*m_maxCorrPerImage);
	hipMalloc(&d_numEntriesPerRow, sizeof(int)*m_maxNumberOfImages);
	hipMalloc(&d_n_match_per_pair, sizeof(int)*m_maxNumberOfImages*m_maxNumberOfImages);
	hipMalloc(&d_update_pose_flags, sizeof(int)*update_pose_flags.size());
	hipMemcpy(d_update_pose_flags, update_pose_flags.data(), sizeof(int)*update_pose_flags.size(), hipMemcpyHostToDevice);

	hipMalloc(&m_solverState.d_countHighResidual, sizeof(int));

	hipMalloc(&m_solverState.d_denseJtJ, sizeof(float) * 36 * numberOfVariables * numberOfVariables);
	hipMalloc(&m_solverState.d_denseJtr, sizeof(float) * 6 * numberOfVariables);
	m_maxNumDenseImPairs = m_maxNumberOfImages * (m_maxNumberOfImages - 1) / 2;
	hipMalloc(&m_solverState.d_denseCorrCounts, sizeof(float) * m_maxNumDenseImPairs);
	hipMalloc(&m_solverState.d_denseOverlappingImages, sizeof(uint2) * m_maxNumDenseImPairs);
	hipMalloc(&m_solverState.d_numDenseOverlappingImages, sizeof(int));

	hipMalloc(&m_solverState.d_corrCount, sizeof(int));
	hipMalloc(&m_solverState.d_corrCountColor, sizeof(int));
	hipMalloc(&m_solverState.d_sumResidualColor, sizeof(float));
	hipMalloc(&m_solverState.d_xTransforms, sizeof(float4x4)*m_maxNumberOfImages);
	hipMalloc(&m_solverState.d_xTransformInverses, sizeof(float4x4)*m_maxNumberOfImages);

#ifdef NEW_GUIDED_REMOVE
	hipMalloc(&d_transforms, sizeof(float4x4)*m_maxNumberOfImages);
#endif

	//solve params
	m_maxResidualThresh = 100;
	m_defaultParams.denseDistThresh = (*yml)["p2p"]["max_dist"].as<float>();
	m_defaultParams.denseNormalThresh = std::cos((*yml)["p2p"]["max_normal_angle"].as<float>()/180.0*M_PI);
	m_defaultParams.sparse_dist_thres = (*yml)["bundle"]["feature_edge_dist_thres"].as<float>();
	m_defaultParams.sparse_normal_thres = std::cos((*yml)["bundle"]["feature_edge_normal_thres"].as<float>()/180.0*M_PI);
	m_defaultParams.icp_pose_rot_thres = (*yml)["bundle"]["icp_pose_rot_thres"].as<float>()/180.0*M_PI;
	m_defaultParams.denseColorThresh = 0.1;
	m_defaultParams.denseColorGradientMin = 0.005;
	m_defaultParams.denseDepthMin = 0.1;
	m_defaultParams.denseDepthMax = 9999;
	m_defaultParams.denseOverlapCheckSubsampleFactor = 1;

#ifdef DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

	//!!!DEBUGGING
	hipMemset(m_solverState.d_deltaRot, 0, sizeof(float3)*numberOfVariables);
	hipMemset(m_solverState.d_deltaTrans, 0, sizeof(float3)*numberOfVariables);
	hipMemset(m_solverState.d_rRot, 0, sizeof(float3)*numberOfVariables);
	hipMemset(m_solverState.d_rTrans, 0, sizeof(float3)*numberOfVariables);
	hipMemset(m_solverState.d_zRot, 0, sizeof(float3)*numberOfVariables);
	hipMemset(m_solverState.d_zTrans, 0, sizeof(float3)*numberOfVariables);
	hipMemset(m_solverState.d_pRot, 0, sizeof(float3)*numberOfVariables);
	hipMemset(m_solverState.d_pTrans, 0, sizeof(float3)*numberOfVariables);
	hipMemset(m_solverState.d_Jp, 0, sizeof(float3)*maxNumResiduals);
	hipMemset(m_solverState.d_Ap_XRot, 0, sizeof(float3)*numberOfVariables);
	hipMemset(m_solverState.d_Ap_XTrans, 0, sizeof(float3)*numberOfVariables);
	hipMemset(m_solverState.d_scanAlpha, 0, sizeof(float) * 2);
	hipMemset(m_solverState.d_rDotzOld, -1, sizeof(float) *numberOfVariables);
	hipMemset(m_solverState.d_precondionerRot, 1, sizeof(float3)*numberOfVariables);
	hipMemset(m_solverState.d_precondionerTrans, 1, sizeof(float3)*numberOfVariables);
	hipMemset(m_solverState.d_sumResidual, 0, sizeof(float));
	hipMemset(m_solverExtra.d_maxResidual, 0, sizeof(float) * n);
	hipMemset(m_solverExtra.d_maxResidualIndex, 0, sizeof(int) * n);
	hipMemset(d_variablesToCorrespondences, 0, sizeof(int)*m_maxNumberOfImages*m_maxCorrPerImage);
	hipMemset(d_numEntriesPerRow, 0, sizeof(int)*m_maxNumberOfImages);
	hipMemset(d_n_match_per_pair, 0, sizeof(int)*m_maxNumberOfImages*m_maxNumberOfImages);
	hipMemset(m_solverState.d_countHighResidual, -1, sizeof(int));
	hipMemset(m_solverState.d_denseJtJ, 0, sizeof(float) * 36 * numberOfVariables * numberOfVariables);
	hipMemset(m_solverState.d_denseJtr, 0, sizeof(float) * 6 * numberOfVariables);
	hipMemset(m_solverState.d_denseCorrCounts, -1, sizeof(float) * m_maxNumDenseImPairs);
	hipMemset(m_solverState.d_denseOverlappingImages, -1, sizeof(uint2) * m_maxNumDenseImPairs);
	hipMemset(m_solverState.d_numDenseOverlappingImages, -1, sizeof(int));

	hipMemset(m_solverState.d_corrCount, -1, sizeof(int));
	hipMemset(m_solverState.d_corrCountColor, -1, sizeof(int));
	hipMemset(m_solverState.d_sumResidualColor, -1, sizeof(float));

#ifdef DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
	//!!!DEBUGGING
}

CUDASolverBundling::~CUDASolverBundling()
{
	if (m_timer) delete m_timer;

	// State
	cutilSafeCall(hipFree(m_solverState.d_deltaRot));
	cutilSafeCall(hipFree(m_solverState.d_deltaTrans));
	cutilSafeCall(hipFree(m_solverState.d_rRot));
	cutilSafeCall(hipFree(m_solverState.d_rTrans));
	cutilSafeCall(hipFree(m_solverState.d_zRot));
	cutilSafeCall(hipFree(m_solverState.d_zTrans));
	cutilSafeCall(hipFree(m_solverState.d_pRot));
	cutilSafeCall(hipFree(m_solverState.d_pTrans));
	cutilSafeCall(hipFree(m_solverState.d_Jp));
	cutilSafeCall(hipFree(m_solverState.d_Ap_XRot));
	cutilSafeCall(hipFree(m_solverState.d_Ap_XTrans));
	cutilSafeCall(hipFree(m_solverState.d_scanAlpha));
	cutilSafeCall(hipFree(m_solverState.d_rDotzOld));
	cutilSafeCall(hipFree(m_solverState.d_precondionerRot));
	cutilSafeCall(hipFree(m_solverState.d_precondionerTrans));
	cutilSafeCall(hipFree(m_solverState.d_sumResidual));
	cutilSafeCall(hipFree(m_solverExtra.d_maxResidual));
	cutilSafeCall(hipFree(m_solverExtra.d_maxResidualIndex));
	SAFE_DELETE_ARRAY(m_solverExtra.h_maxResidual);
	SAFE_DELETE_ARRAY(m_solverExtra.h_maxResidualIndex);

	cutilSafeCall(hipFree(d_variablesToCorrespondences));
	cutilSafeCall(hipFree(d_numEntriesPerRow));
	cutilSafeCall(hipFree(d_n_match_per_pair));
	cutilSafeCall(hipFree(d_update_pose_flags));

	cutilSafeCall(hipFree(m_solverState.d_countHighResidual));
	cutilSafeCall(hipFree(m_solverState.d_denseCorrCounts));
	cutilSafeCall(hipFree(m_solverState.d_denseJtJ));
	cutilSafeCall(hipFree(m_solverState.d_denseJtr));

	cutilSafeCall(hipFree(m_solverState.d_xTransforms));
	cutilSafeCall(hipFree(m_solverState.d_xTransformInverses));
	cutilSafeCall(hipFree(m_solverState.d_denseOverlappingImages));
	cutilSafeCall(hipFree(m_solverState.d_numDenseOverlappingImages));

	cutilSafeCall(hipFree(m_solverState.d_corrCount));
	cutilSafeCall(hipFree(m_solverState.d_sumResidualColor));
	cutilSafeCall(hipFree(m_solverState.d_corrCountColor));

#ifdef NEW_GUIDED_REMOVE
	cutilSafeCall(hipFree(d_transforms));
#endif
}

void CUDASolverBundling::solve(EntryJ* d_correspondences, unsigned int numberOfCorrespondences, const int* d_validImages, unsigned int numberOfImages,const CUDACache* cudaCache, const std::vector<float>& weightsSparse, const std::vector<float>& weightsDenseDepth, const std::vector<float>& weightsDenseColor, bool usePairwiseDense, float3* d_rotationAnglesUnknowns, float3* d_translationUnknowns, bool rebuildJT, bool findMaxResidual, unsigned int revalidateIdx)
{
	unsigned int nNonLinearIterations = (*yml)["bundle"]["num_iter_outter"].as<int>();
	assert(numberOfImages > 1 && nNonLinearIterations > 0);
	if (numberOfCorrespondences > m_maxCorrPerImage*m_maxNumberOfImages) {
		//warning: correspondences will be invalidated AT RANDOM!
		std::cerr << "WARNING: #corr (" << numberOfCorrespondences << ") exceeded limit (" << m_maxCorrPerImage << "*" << m_maxNumberOfImages << "), please increase max #corr per image in the GAS" << std::endl;
	}

	float* convergence = NULL;
	if (m_bRecordConvergence) {
		m_convergence.resize(nNonLinearIterations + 1, -1.0f);
		convergence = m_convergence.data();
	}

	m_solverState.d_xRot = d_rotationAnglesUnknowns;
	m_solverState.d_xTrans = d_translationUnknowns;

	SolverParameters parameters = m_defaultParams;
	parameters.nNonLinearIterations = nNonLinearIterations;
	parameters.nLinIterations = (*yml)["bundle"]["num_iter_inner"].as<int>();;
	parameters.verifyOptDistThresh = m_verifyOptDistThresh;
	parameters.verifyOptPercentThresh = m_verifyOptPercentThresh;
	parameters.highResidualThresh = std::numeric_limits<float>::infinity();
	parameters.robust_delta = (*yml)["bundle"]["robust_delta"].as<float>();
	parameters.depth_association_radius = (*yml)["bundle"]["depth_association_radius"].as<int>();

	parameters.weightSparse = weightsSparse.front();
	parameters.weightDenseDepth = weightsDenseDepth.front();
	parameters.weightDenseColor = weightsDenseColor.front();
	parameters.useDense = (parameters.weightDenseDepth > 0 || parameters.weightDenseColor > 0);
	parameters.useDenseDepthAllPairwise = usePairwiseDense;

	SolverInput solverInput;
	solverInput.d_correspondences = d_correspondences;
	solverInput.d_variablesToCorrespondences = d_variablesToCorrespondences;
	solverInput.d_numEntriesPerRow = d_numEntriesPerRow;
	solverInput.numberOfImages = numberOfImages;
	solverInput.numberOfCorrespondences = numberOfCorrespondences;
	solverInput.d_n_match_per_pair = d_n_match_per_pair;
	solverInput.d_update_pose_flags = d_update_pose_flags;
	solverInput.maxNumberOfImages = m_maxNumberOfImages;
	solverInput.maxCorrPerImage = m_maxCorrPerImage;
	solverInput.maxNumDenseImPairs = m_maxNumDenseImPairs;

	solverInput.weightsSparse = weightsSparse.data();
	solverInput.weightsDenseDepth = weightsDenseDepth.data();
	solverInput.weightsDenseColor = weightsDenseColor.data();
	solverInput.d_validImages = d_validImages;
	solverInput.d_cacheFrames = cudaCache->getCacheFramesGPU();
	solverInput.denseDepthWidth = cudaCache->getWidth(); //TODO constant buffer for this?
	solverInput.denseDepthHeight = cudaCache->getHeight();
	Eigen::Matrix4f intrinsics = cudaCache->getIntrinsics();
	solverInput.intrinsics = make_float4(intrinsics(0, 0), intrinsics(1, 1), intrinsics(0, 2), intrinsics(1, 2));
	assert(solverInput.denseDepthWidth / parameters.denseOverlapCheckSubsampleFactor > 8); //need enough samples


#ifdef NEW_GUIDED_REMOVE
	convertLiePosesToMatricesCU(m_solverState.d_xRot, m_solverState.d_xTrans, solverInput.numberOfImages, d_transforms, m_solverState.d_xTransformInverses); //debugging only (store transforms before opt)
#endif
#ifdef DEBUG_PRINT_SPARSE_RESIDUALS
	if (findMaxResidual) {
		float residualBefore = EvalResidual(solverInput, m_solverState, parameters, NULL);
		computeMaxResidual(solverInput, parameters, (unsigned int)-1);
		Eigen::Vector<uint, 2> beforeMaxImageIndices;
    float beforeMaxRes;
    unsigned int curFrame = (revalidateIdx == (unsigned int)-1) ? solverInput.numberOfImages - 1 : revalidateIdx;
		getMaxResidual(curFrame, d_correspondences, beforeMaxImageIndices, beforeMaxRes);
		std::cout << "\tbefore: (" << solverInput.numberOfImages << ") sumres = " << residualBefore << " / " << solverInput.numberOfCorrespondences << " = " << residualBefore / (float)solverInput.numberOfCorrespondences << " | maxres = " << beforeMaxRes << " images (" << beforeMaxImageIndices << ")" << std::endl;
	}
#endif


	if (rebuildJT)
	{
		buildVariablesToCorrespondencesTable(d_correspondences, numberOfCorrespondences);
	}

#ifdef DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

	solveBundlingStub(solverInput, m_solverState, parameters, m_solverExtra, convergence, m_timer);

	if (findMaxResidual) {
		computeMaxResidual(solverInput, parameters, revalidateIdx);
#ifdef DEBUG_PRINT_SPARSE_RESIDUALS
		float residualAfter = EvalResidual(solverInput, m_solverState, parameters, NULL);
		Eigen::Vector<uint, 2> afterMaxImageIndices;
    float afterMaxRes;
    unsigned int curFrame = (revalidateIdx == (unsigned int)-1) ? solverInput.numberOfImages - 1 : revalidateIdx;
		getMaxResidual(curFrame, d_correspondences, afterMaxImageIndices, afterMaxRes);
		std::cout << "\tafter: (" << solverInput.numberOfImages << ") sumres = " << residualAfter << " / " << solverInput.numberOfCorrespondences << " = " << residualAfter / (float)solverInput.numberOfCorrespondences << " | maxres = " << afterMaxRes << " images (" << afterMaxImageIndices << ")" << std::endl;
#endif
	}
}

void CUDASolverBundling::buildVariablesToCorrespondencesTable(EntryJ* d_correspondences, unsigned int numberOfCorrespondences)
{
	cutilSafeCall(hipMemset(d_numEntriesPerRow, 0, sizeof(int)*m_maxNumberOfImages));

	if (numberOfCorrespondences > 0)
		buildVariablesToCorrespondencesTableCUDA(d_correspondences, numberOfCorrespondences, m_maxCorrPerImage, d_variablesToCorrespondences, d_numEntriesPerRow, m_timer, d_n_match_per_pair, m_maxNumberOfImages);
}

void CUDASolverBundling::computeMaxResidual(SolverInput& solverInput, SolverParameters& parameters, unsigned int revalidateIdx)
{
	if (m_timer) m_timer->startEvent(__FUNCTION__);
	if (parameters.weightSparse > 0.0f) {
		evalMaxResidual(solverInput, m_solverState, m_solverExtra, parameters, NULL);//m_timer);
		// copy to cpu
		unsigned int n = (solverInput.numberOfCorrespondences + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
		cutilSafeCall(hipMemcpy(m_solverExtra.h_maxResidual, m_solverExtra.d_maxResidual, sizeof(float) * n, hipMemcpyDeviceToHost));
		cutilSafeCall(hipMemcpy(m_solverExtra.h_maxResidualIndex, m_solverExtra.d_maxResidualIndex, sizeof(int) * n, hipMemcpyDeviceToHost));
		// compute max
		float maxResidual = 0.0f; int maxResidualIndex = 0;
		for (unsigned int i = 0; i < n; i++) {
			if (maxResidual < m_solverExtra.h_maxResidual[i]) {
				maxResidual = m_solverExtra.h_maxResidual[i];
				maxResidualIndex = m_solverExtra.h_maxResidualIndex[i];
			}
		}
#ifdef NEW_GUIDED_REMOVE

		//if (solverInput.numberOfImages == 51) {
		//	SensorData sd; sd.loadFromFile("../data/iclnuim/aliv2.sens");
		//	std::vector<mat4f> trajectory(solverInput.numberOfImages);
		//	hipMemcpy(trajectory.data(), d_transforms, sizeof(mat4f)*trajectory.size(), hipMemcpyDeviceToHost));
		//	sd.saveToPointCloud("debug/tmp.ply", trajectory, 0, solverInput.numberOfImages*10, 10, true);
		//	int a = 5;
		//}

		m_maxResImPairs.clear();
		if (maxResidual > GUIDED_SEARCH_MAX_RES_THRESH) {
			parameters.highResidualThresh = std::min(std::max(0.2f * maxResidual, 0.1f), 0.4f);
			collectHighResiduals(solverInput, m_solverState, m_solverExtra, parameters, m_timer);
			unsigned int highResCount;
			cutilSafeCall(hipMemcpy(&highResCount, m_solverState.d_countHighResidual, sizeof(unsigned int), hipMemcpyDeviceToHost));
			n = std::min(highResCount, (m_maxCorrPerImage*m_maxNumberOfImages + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
			cutilSafeCall(hipMemcpy(m_solverExtra.h_maxResidual, m_solverExtra.d_maxResidual, sizeof(float) * n, hipMemcpyDeviceToHost));
			cutilSafeCall(hipMemcpy(m_solverExtra.h_maxResidualIndex, m_solverExtra.d_maxResidualIndex, sizeof(int) * n, hipMemcpyDeviceToHost));

			if (n > 1) {
				// check high residuals with previous trajectory as reference //TODO MAKE EFFICIENT
				std::vector<float4x4> transforms(solverInput.numberOfImages);
				hipMemcpy(transforms.data(), d_transforms, sizeof(float4x4)*solverInput.numberOfImages, hipMemcpyDeviceToHost);
				std::unordered_map<vec2ui, float> residualMap; //TODO should be something better than this...
				std::unordered_map<vec2ui, float> allCollectedResidualMap; //debugging
				std::vector<EntryJ> corrs(n);
				for (unsigned int i = 0; i < n; i++) {
					hipMemcpy(corrs.data() + i, solverInput.d_correspondences + m_solverExtra.h_maxResidualIndex[i], sizeof(EntryJ), hipMemcpyDeviceToHost);
					const EntryJ& h_corr = corrs[i];
					Eigen::Vector<uint, 2> imageIndices(h_corr.imgIdx_i, h_corr.imgIdx_j);
					//compute res at previous
					if (h_corr.imgIdx_j == solverInput.numberOfImages - 1 && std::abs((int)h_corr.imgIdx_i - (int)h_corr.imgIdx_j) > 10) { //introduced by latest image
						float3 prevRes = fabs(transforms[h_corr.imgIdx_i] * h_corr.pos_i - transforms[h_corr.imgIdx_j] * h_corr.pos_j); //eval new corrs with previous trajectory
						float prevMaxRes = fmaxf(prevRes.z, fmaxf(prevRes.x, prevRes.y));
						if (prevMaxRes > 1.5f*m_solverExtra.h_maxResidual[i]) {
							auto it = residualMap.find(imageIndices);
							if (it == residualMap.end()) residualMap[imageIndices] = m_solverExtra.h_maxResidual[i];
							else it->second = std::max(m_solverExtra.h_maxResidual[i], it->second);
						}
					}
					else if (h_corr.imgIdx_j == revalidateIdx && std::abs((int)h_corr.imgIdx_i - (int)h_corr.imgIdx_j) > 10) { //introduced by latest revalidate
						auto it = residualMap.find(imageIndices);
						if (it == residualMap.end()) residualMap[imageIndices] = m_solverExtra.h_maxResidual[i];
						else it->second = std::max(m_solverExtra.h_maxResidual[i], it->second);
					}
					auto it = allCollectedResidualMap.find(imageIndices);
					if (it == allCollectedResidualMap.end()) allCollectedResidualMap[imageIndices] = m_solverExtra.h_maxResidual[i];
					else it->second = std::max(m_solverExtra.h_maxResidual[i], it->second);
				}
				if (!residualMap.empty()) { //debug print
					unsigned int rep = residualMap.begin()->first.x;
					std::cout << "rep: (" << rep << ", " << solverInput.numberOfImages - 1 << ")" << std::endl;
					for (const auto& r : residualMap) m_maxResImPairs.push_back(r.first);

					////one extra solve
					//parameters.nNonLinearIterations = 1;
					//solveBundlingStub(solverInput, m_solverState, parameters, m_solverExtra, NULL, m_timer);

					////!!!debugging
					//{
					//	static SensorData sd;
					//	if (sd.m_frames.empty()) sd.loadFromFile("../data/iclnuim/aliv2.sens");
					//	std::vector<mat4f> trajectory(solverInput.numberOfImages);
					//	hipMemcpy(trajectory.data(), d_transforms, sizeof(mat4f)*trajectory.size(), hipMemcpyDeviceToHost));
					//	sd.saveToPointCloud("debug/tmp/" + std::to_string(solverInput.numberOfImages) + "-init.ply", trajectory, 0, solverInput.numberOfImages*10, 10, true);
					//	convertLiePosesToMatricesCU(m_solverState.d_xRot, m_solverState.d_xTrans, solverInput.numberOfImages, d_transforms, m_solverState.d_xTransformInverses);
					//	hipMemcpy(trajectory.data(), d_transforms, sizeof(mat4f)*trajectory.size(), hipMemcpyDeviceToHost));
					//	sd.saveToPointCloud("debug/tmp/" + std::to_string(solverInput.numberOfImages) + "-opt.ply", trajectory, 0, solverInput.numberOfImages*10, 10, true);
					//	int a = 5;
					//}
					////!!!debugging
				}
			}
		}
#endif
		m_solverExtra.h_maxResidual[0] = maxResidual;
		m_solverExtra.h_maxResidualIndex[0] = maxResidualIndex;
	}
	else {
		m_solverExtra.h_maxResidual[0] = 0.0f;
		m_solverExtra.h_maxResidualIndex[0] = 0;
	}
	if (m_timer) m_timer->endEvent();
}

bool CUDASolverBundling::getMaxResidual(unsigned int curFrame, EntryJ* d_correspondences, Eigen::Vector<uint, 2>& imageIndices, float& maxRes)
{
	maxRes = m_solverExtra.h_maxResidual[0];

	// for debugging get image indices regardless
	EntryJ h_corr;
	unsigned int imIdx = m_solverExtra.h_maxResidualIndex[0];
	cutilSafeCall(hipMemcpy(&h_corr, d_correspondences + imIdx, sizeof(EntryJ), hipMemcpyDeviceToHost));
	imageIndices = Eigen::Vector<uint, 2>(h_corr.imgIdx_i, h_corr.imgIdx_j);

	bool remove = false;
	//const float curThresh = (imageIndices.y == curFrame) ? m_maxResidualThresh : m_maxResidualThresh * 2.0f; //TODO try this out
	const float curThresh = m_maxResidualThresh;
	if (!(imageIndices(0) == 0 && imageIndices(1) < 10) && m_solverExtra.h_maxResidual[0] > curThresh) remove = true; //don't remove the first frame

	//!!!debugging //TODO REMOVE THIS
	if (m_solverExtra.h_maxResidual[0] > curThresh && imageIndices(0) == 0 && imageIndices(1) < 10) {
		std::cout << "warning! max residual would invalidate images " << imageIndices.transpose() << " (" << m_solverExtra.h_maxResidual[0] << ")" << std::endl;
		//getchar();
	}
	//!!!debugging

	return remove;
}

bool CUDASolverBundling::useVerification(EntryJ* d_correspondences, unsigned int numberOfCorrespondences)
{
	SolverParameters parameters;
	parameters.nNonLinearIterations = 0;
	parameters.nLinIterations = 0;
	parameters.verifyOptDistThresh = m_verifyOptDistThresh;
	parameters.verifyOptPercentThresh = m_verifyOptPercentThresh;

	SolverInput solverInput;
	solverInput.d_correspondences = d_correspondences;
	solverInput.d_variablesToCorrespondences = NULL;
	solverInput.d_numEntriesPerRow = NULL;
	solverInput.numberOfImages = 0;
	solverInput.numberOfCorrespondences = numberOfCorrespondences;

	solverInput.maxNumberOfImages = m_maxNumberOfImages;
	solverInput.maxCorrPerImage = m_maxCorrPerImage;

	unsigned int numHighResiduals = countHighResiduals(solverInput, m_solverState, parameters, m_timer);
	//std::cout << "\t[ useVerification ] " << numHighResiduals << " / " << solverInput.numberOfCorrespondences << " = " << (float)numHighResiduals / solverInput.numberOfCorrespondences << " vs " << parameters.verifyOptPercentThresh << std::endl;
	if ((float)numHighResiduals / solverInput.numberOfCorrespondences >= parameters.verifyOptPercentThresh) return true;
	return false;
}
